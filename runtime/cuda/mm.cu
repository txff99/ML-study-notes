#include <hip/hip_runtime.h>
#include <stdio.h>
#ifdef USE_CUTLASS
#include <cutlass_helper.h>
#endif

extern "C" {

__global__ void matmul_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < M && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < K; ++k) {
            sum += A[row * K + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

__global__ void add_kernel(const float* A, const float* B, float* C, int M, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < M && col < N) {
        C[row * N + col] = A[row * N + col] + B[row * N + col];
    }
}

__global__ void sub_kernel(const float* A, const float* B, float* C, int M, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < M && col < N) {
        C[row * N + col] = A[row * N + col] - B[row * N + col];
    }
}

__global__ void expand_kernel(const float* A, float* B, int M, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < M && col < N) {
        B[row * N + col] = A[col];
    }
}

void matmul(const float* A, const float* B, float* C, int M, int N, int K) {
    dim3 block(32, 16);
    dim3 grid((N + block.x - 1) / block.x, (M + block.y - 1) / block.y);
    matmul_kernel<<<grid, block>>>(A, B, C, M, N, K);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch error: %s\n", hipGetErrorString(err));
    }
}
#ifdef USE_CUTLASS
void cutlass_mma(const float* A, const float* B, float* C, int M, int N, int K){
    float alpha = 1;
    float beta = 0;
    int lda = K;
    int ldb = N;
    int ldc = N;
    auto result = CutlassSgemmNN(N, M, K, alpha, B, ldb, A, lda, beta, C, ldc);

  if (result != cudaSuccess) {
    std::cerr << "CUTLASS GEMM kernel failed: "
      << cudaGetErrorString(result) << std::endl;
  }
}
#else
void cutlass_mma(const float* A, const float* B, float* C, int M, int N, int K){
    fprintf(stderr,  "CUTLASS NOT ENABLED, use USE_CUTLASS=1 during build\n");
}
#endif

void add(const float* A, const float* B, float* C, int M, int N){
    dim3 block(32, 16);
    dim3 grid((N + block.x - 1) / block.x, (M + block.y - 1) / block.y);

    add_kernel<<<grid, block>>>(A, B, C, M, N);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch error: %s\n", hipGetErrorString(err));
    }
}

void sub(const float* A, const float* B, float* C, int M, int N){
    dim3 block(32, 16);
    dim3 grid((N + block.x - 1) / block.x, (M + block.y - 1) / block.y);

    sub_kernel<<<grid, block>>>(A, B, C, M, N);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch error: %s\n", hipGetErrorString(err));
    }
}

void expand(const float* A, float* B, int M, int N){
    dim3 block(32, 16);
    dim3 grid((N + block.x - 1) / block.x, (M + block.y - 1) / block.y);

    expand_kernel<<<grid, block>>>(A, B, M, N);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch error: %s\n", hipGetErrorString(err));
    }
}
}
