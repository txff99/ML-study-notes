#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" void* alloc_on_gpu(int size_bytes) {
    void* d_ptr = nullptr;
    hipError_t err = hipMalloc(&d_ptr, size_bytes);
    if (err != hipSuccess) {
        printf("hipMalloc failed: %s\n", hipGetErrorString(err));
        return nullptr; 
    }
    return d_ptr;
}

extern "C" void free_on_gpu(void* ptr) {
    hipError_t err = hipFree(ptr);
    if (err != hipSuccess) {
        printf("hipFree failed: %s\n", hipGetErrorString(err));
    }
}

extern "C" void cuda_copy_to_device(void* dst, const void* src, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
}

extern "C" void cuda_copy_to_host(void* dst, const void* src, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
}