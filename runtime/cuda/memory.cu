#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" void* alloc_on_gpu(int size_bytes) {
    void* d_ptr = nullptr;
    hipError_t err = hipMalloc(&d_ptr, size_bytes);
    if (err != hipSuccess) {
        printf("hipMalloc failed: %s\n", hipGetErrorString(err));
        return nullptr; 
    }
    return d_ptr;
}

extern "C" void free_on_gpu(void* ptr) {
    hipFree(ptr);
    if (err != hipSuccess) {
        printf("hipFree failed: %s\n", hipGetErrorString(err));
    }
}

void cuda_copy_to_device(void* dst, const void* src, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
}

void cuda_copy_to_host(void* dst, const void* src, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
}